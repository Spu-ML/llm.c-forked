#include "hip/hip_runtime.h"
/*
Kernels for copy & transpose with format conversion (+ optional elementwise operations, e.g. GELU)
Many parameters are configurable by changing the defines

Compile examples (change 90 to your SM architecture - do not trust performance without it):
nvcc -O3 --generate-code arch=compute_90,code=[compute_90,sm_90] --use_fast_math advanced_copy_transpose.cu -o advanced_copy_transpose
nvcc -DENABLE_GELU -DIN_TYPE=half -DOUT_TYPE=float -DSCALING_FACTOR=0.5f -DTRANSPOSE_AND_COPY=true -O3 --generate-code arch=compute_90,code=[compute_90,sm_90] --use_fast_math advanced_copy_transpose.cu -o advanced_copy_transpose

Useful defines (not all options available in all kernels):

IN_TYPE=float (input data type, default is __hip_bfloat16)
OUT_TYPE=half (output data type, default is __hip_fp8_e4m3_fnuz)
SCALING_FACTOR=0.5f (scaling factor for the output, default is no scaling i.e. 1.0f)
TRANSPOSE_AND_COPY=true (enable extra converted copy of the input tensor for transpose kernels, default is false)
ENABLE_GELU=true (enable GELU elementwise function, default is false)
CALCULATE_ABSMAX=true (calculate absmax of the output tensor pre-scaling, default is false)
ABSMAX_EXPONENT_ONLY=true (round absmax down by clearing all the mantissa bits, default is false)
DEFAULT_TILE=64 (tile size for transpose kernels, this affects shared memory and maximum block size, default=32)
WIDTH=8192 (width of the input tensor, default=8192)
HEIGHT=3072 (height of the input tensor, default=3072)
ABSMAX_ITERATIONS_PER_THREAD=2 (outer loop iterations for absmax kernel 20)
FUSED_ABSMAX_FIRST_PHASE_BYTES=52428800 (number of bytes the fused absmax assumes can fit in the L2 cache)
FUSED_RESCALE_IN_PLACE=true (always rescale to avoid all overflow in the fused absmax kernel 30)

Kernel versions:

version 0 is a non-optimized copy (not a transpose)
version 1 is a simple fast copy similar to version 3, but without all the extra functionality (GELU/absmax/scaling/etc.)
version 2 is a highly optimized copy that tries to keep all loads/stores 128-bit
version 3 is a simpler very optimized copy (with support for absmax calculation)

version 10 is a non-optimized transpose (no elementwise, no absmax)
version 11 is a fast transpose with shared memory (no support for absmax at the moment)
version 12 is a very fast transpose with shared memory and 128-bit loads/stores (with support for absmax calculation)

version 20 is an absmax-only kernel (no copy or transpose)
version 21 is the same but without the memset (sanity checking, ~no performance difference)
version 22 is the same as 20 but with persistent threads

version 25 is a non-fused absmax + scale with 2 kernel invocations
version 26 is the same as 25 but the copy is in reverse order to maximise L2 cache hits
version 30 is an extremely complicated fused absmax+scale+rescale kernel (see defines above)

Usage example: ./advanced_copy_transpose 12
*/

#define SKIP_CUBLAS // to save compile time
#include "common.h"
#include <cstring>
#include <hip/hip_fp8.h>

//#define IN_TYPE half
//#define OUT_TYPE __hip_fp8_e5m2_fnuz
//#define SCALING_FACTOR 0.3f
//#define TRANSPOSE_AND_COPY true
//#define ENABLE_GELU true
//#define CALCULATE_ABSMAX true
//#define ABSMAX_EXPONENT_ONLY true
//#define DEFAULT_TILE 64UL
//#define WIDTH 8192
//#define HEIGHT 768
//#define ABSMAX_ITERATIONS_PER_THREAD 1
//#define FUSED_ABSMAX_FIRST_PHASE_BYTES 52428800
//#define FUSED_RESCALE_IN_PLACE true

#if !defined(IN_TYPE)
#define IN_TYPE __hip_fp8_e4m3_fnuz
#endif
#if !defined(OUT_TYPE)
#define OUT_TYPE __hip_fp8_e4m3_fnuz
#endif

#if defined(SCALING_FACTOR)
#define SCALING true
#else
#define SCALING_FACTOR 1.0f
#define SCALING false
#endif

#if !defined(TRANSPOSE_AND_COPY)
#define TRANSPOSE_AND_COPY false
#endif
#if !defined(ENABLE_GELU)
#define ENABLE_GELU false
#endif

#if !defined(CALCULATE_ABSMAX)
#define CALCULATE_ABSMAX false
#endif
#define DEFAULT_ABSMAX_FACTOR 2
#if CALCULATE_ABSMAX == true
#define ABSMAX_FACTOR DEFAULT_ABSMAX_FACTOR
#else
#define ABSMAX_FACTOR 0
#endif
#if !defined(ABSMAX_EXPONENT_ONLY)
#define ABSMAX_EXPONENT_ONLY false
#endif

#if !defined(DEFAULT_TILE)
#define DEFAULT_TILE 32UL // 32x32 transpose is a good default but 64x64 might be better for absmax
#endif
#if !defined(WIDTH)
#define WIDTH 768
#endif
#if !defined(HEIGHT)
#define HEIGHT 50304
#endif

#if !defined(ABSMAX_ITERATIONS_PER_THREAD)
#define ABSMAX_ITERATIONS_PER_THREAD 2
#endif

#if !defined(FUSED_ABSMAX_FIRST_PHASE_BYTES)
#define FUSED_ABSMAX_FIRST_PHASE_BYTES (40*1024*1024)
#endif

#if !defined(FUSED_RESCALE_IN_PLACE)
#define FUSED_RESCALE_IN_PLACE false
#endif

#if !defined(DEFAULT_Z_DIM)
#define DEFAULT_Z_DIM 2
#endif

// ----------------------------------------------------------------------------
// these are passed as default kernel parameters to avoid making everything too messy
unsigned int* d_absmax_estimate = NULL;
unsigned int* d_absmax_counter = NULL;
unsigned int* d_absmax_actual = NULL;
unsigned int absmax_storage = 0;
float* d_scaling_factor = NULL;

// misc. useful constants
constexpr int FIRST_TRANSPOSE_KERNEL = 10; // kernels 0/1/2/3 are copy kernels without transpose
constexpr int FIRST_ABSMAX_ONLY_KERNEL = 20; // kernels 20+ are absmax kernels, they do not copy or transpose

// -----./-----------------------------------------------------------------------
// elementwise functions which can be applied as part of the copy/transpose
// for elementwise kernels that require metadata (e.g. layernorm forward with known mean/std),
// we could maybe store it in constant buffers rather than in yet-another-function-parameter...
using elementwise_func_t = float (*) (float, const void**, uint, uint, uint, uint);
#if ENABLE_GELU == true
#define DEFAULT_ELEMENTWISE gelu_forward_elementwise
#else
#define DEFAULT_ELEMENTWISE nothing_elementwise
#endif

__host__ __device__ float nothing_elementwise(float in, const void** __restrict__ metadata, uint x, uint y, uint w, uint h) {
    (void)x; (void)y; (void)w; (void)h; (void)metadata;
    return in;
}

__host__ __device__ float gelu_forward_elementwise(float in, const void** __restrict__ metadata, uint x, uint y, uint w, uint h) {
    (void)x; (void)y; (void)w; (void)h; (void)metadata;
    float cube = 0.044715f * in * in * in;

    float tanh_out;
    float tanh_arg = sqrtf(2.0f / M_PI) * (in + cube);
    // only use asm if this is the device
#if defined(__CUDA_ARCH__)
    asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_out) : "f"(tanh_arg));
#else
    tanh_out = tanhf(tanh_arg);
#endif

    // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
    float half_in = 0.5f * in;
    return half_in * tanh_out + half_in;
}

// ----------------------------------------------------------------------------
// CPU code reference

template <bool scaling=SCALING, typename T1, typename T2>
void transpose_cpu(T1* transposed, T1* transposed_gelu, T1* copy, T1* copy_gelu,
                   const T2* input, size_t width, size_t height, float scaling_factor=SCALING_FACTOR, const void** metadata=NULL) {
    for (size_t y = 0; y < height; y++) {
        for (size_t x = 0; x < width; x++) {
            // note (IN_TYPE) unlike GPU version because T2 is actually always float for simplicity
            float in = (float)((IN_TYPE)input[x + y*width]);
            float gelu = gelu_forward_elementwise(in, metadata, x, y, width, height);

            // absmax calculation is pre-scaling (but has its own ABSMAX_FACTOR)
            float absmax_factor = (ABSMAX_FACTOR != 0) ? (float)ABSMAX_FACTOR : DEFAULT_ABSMAX_FACTOR;
            #if ENABLE_GELU == true
            float absmax = gelu / (float)absmax_factor;
            #else
            float absmax = in / (float)absmax_factor;
            #endif
            constexpr uint absmax_mask = ABSMAX_EXPONENT_ONLY ? 0x7f800000 : 0x7fffffff;
            absmax_storage = max(absmax_storage, *((uint*)&absmax) & absmax_mask);
            #

            if constexpr (scaling) {
                in *= scaling_factor;
                gelu *= scaling_factor;
            }
            transposed[y + x * height] = (T1)in;
            transposed_gelu[y + x * height] = (T1)gelu;
            copy[x + y*width] = (T1)in;
            copy_gelu[x + y*width] = (T1)gelu;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels
// ----------------------------------------------------------------------------
// This helper is for when we want to copy from e.g. FP32 to BF16
// so if want to load a f128 of 4 elements, and write those 4 elements to memory as 64-bit
// not needed in the case of loads, the compiler will automatically optimise away unused reads
// (we might want to replace this with something like a fixed vector width class though)
template<class OriginalType, class ElementType>
__device__ void store_same_length(ElementType* target, Packed128<ElementType> value) {
    int4 bits = value.get_bits();
    switch (sizeof(OriginalType) / sizeof(ElementType)) {
        case 0: *reinterpret_cast<int4*>(target) = bits; break; // smaller
        case 1: *reinterpret_cast<int4*>(target) = bits; break; // same size
        case 2: *reinterpret_cast<int2*>(target) = make_int2(bits.x, bits.y); break;
        case 4: *reinterpret_cast<int*>(target) = bits.x; break;
        default: break; //assert(false);
    }
}

// updates the absmax for the entire threadgroup
// requires all warps in threadblock to be active
// caller can rely on there always being a __syncthreads() for other work
template <bool is2D=false, bool is3D=false> // templating to avoid useless calculations for 1D
__device__ void update_global_absmax(unsigned int* absmax_output, unsigned int absmax_uint) {
    uint bidY = (is2D ? blockDim.y : 1) * (is3D ? blockDim.z : 1);
    uint tidY = (is2D ? threadIdx.y : 0) + (is3D ? threadIdx.z * blockDim.y : 0);
    uint tidXY = threadIdx.x + blockDim.x * tidY;
    uint num_warps = (blockDim.x * bidY) / 32;
    uint lane_id = tidXY % 32;
    uint warp_id = tidXY / 32;

    // use native integer reductions as much as possible (supported on all GPUs with FP8)
    // todo - we could use cooperative groups instead of PTX here but it'd increase compile time
    asm volatile("redux.sync.max.u32 %0, %0, 0xff;" : "+r"(absmax_uint));
    __shared__ uint tmp[32];
    if (lane_id == 0) {
        tmp[warp_id] = absmax_uint;
    }
    __syncthreads();

    if (warp_id == 0) {
        absmax_uint = tmp[lane_id < num_warps ? lane_id : 0];
        // compiler automatically does a warp reduction here and global atomic is single-threaded
        // if we try to do it ourselves, we might end up with *two* warp reductions :(
        atomicMax(absmax_output, absmax_uint);
    }
}

// absmax factor is related to the maximum value of the target format, e.g. 448 for FP8 e4m3
// we skip everything absmax-related if it is at the default value of 0
// absmax_factor should be a constant known at compile time for performance
template <bool always=false, bool absmax_exponent_only=ABSMAX_EXPONENT_ONLY, typename T>
__device__ void update_local_absmax(unsigned int &absmax_uint, T data, uint absmax_factor=0) {
    if (always || absmax_factor != 0) {
        constexpr uint absmax_mask = absmax_exponent_only ? 0x7f800000 : 0x7fffffff;
        absmax_uint = max(absmax_uint, __float_as_uint((float)data / (float)absmax_factor) & absmax_mask);
    }
}

// ----------------------------------------------------------------------------
// GPU kernels for copy

template <bool scaling=SCALING, typename T1, typename T2>
__global__ void copy_naive_kernel0(T1 *copy, const T2 *input, size_t N, const float* __restrict__ scale_pointer=d_scaling_factor) {
    size_t n = (blockIdx.x * blockDim.x + threadIdx.x);
    if (n >= N) { return; }
    copy[n] = (T1)((float)input[n] * (scaling ? *scale_pointer : 1.0f));
}

// simplified copy & format conversion kernel using store_same_length
// keeps the largest format at 128-bit and smallest at 32-bit or 64-bit
template <bool scaling=SCALING, typename T1, typename T2>
__global__ void copy_fast_kernel1(T1 *copy, const T2 *input, size_t N, const float* __restrict__ scale_pointer=d_scaling_factor) {
    // Calculate the *smallest* of the two vector sizes in terms of elements (both are 128-bit if fully used)
    constexpr size_t vec_size = 16 / ((sizeof(T1) < sizeof(T2)) ? sizeof(T2) : sizeof(T1));
    size_t n = (blockIdx.x * blockDim.x + threadIdx.x) * vec_size;
    if (n >= N) { return; }

    // Scaling factor of 1.0f will automatically be optimised away by the compiler
    float scale_factor = scaling ? *scale_pointer : 1.0f;

    // note: if sizeof(T1) < sizeof(T2), compiler will skip unused elements of load128
    // so it may turn out to be a ldg.32 or ldg.64
    Packed128<T2> inp128 = load128<T2>(input + n);
    Packed128<T1> out128;
    for (int k = 0; k < vec_size; k++) {
        out128[k] = (T1)((float)inp128[k] * scale_factor);
    }

    // if sizeof(T2) < sizeof(T1), this will use stg.32 or stg.64 instead of stg.128
    store_same_length<T2,T1>(copy + n, out128);
}

// overly complicated copy & format conversion kernel without store_same_length
// this keeps all loads & stores 128-bit at the cost of more complexity and more register pressure
template <bool scaling=SCALING, elementwise_func_t elementwise_func=DEFAULT_ELEMENTWISE,
          uint absmax_factor=ABSMAX_FACTOR, typename T1, typename T2>
__global__ void copy_advanced_kernel2(T1 *copy, const T2 *input, size_t N, const float* __restrict__ scale_pointer=d_scaling_factor, unsigned int* absmax_output=d_absmax_estimate, const void** meta=NULL) {
    // Optional fused absmax calculation
    uint absmax_uint = 0;

    size_t n = (blockIdx.x * blockDim.x + threadIdx.x) * Packed128<T1>::size;
    if (n >= N) { return; }

    // note: if sizeof(T1) < sizeof(T2), compiler will skip unused elements of load128
    // so it may turn out to be a load32 or load64
    Packed128<T2> inp128;
    Packed128<T1> out128;
    float scale_factor = scaling ? *scale_pointer : 1.0f;
    for (int o = 0; o < max(1, out128.size/inp128.size); o++) {
        inp128 = load128cs<T2>(input + n + o*inp128.size);
        for (int k = 0; k < min(inp128.size, out128.size); k++) {
            float out_float = elementwise_func((float)inp128[k], meta, n+o*inp128.size, 0, N, 1);
            out128[k+o*inp128.size] = (T1)(out_float * (scaling ? scale_factor : 1.0f));
            update_local_absmax(absmax_uint, out_float, absmax_factor); // optional absmax
        }
    }
    store128<T1>(copy + n, out128);

    // update absmax if required
    if constexpr (absmax_factor != 0) {
        update_global_absmax<false>(absmax_output, absmax_uint);
    }
}

// simplified copy & format conversion kernel using store_same_length
// keeps the largest format at 128-bit and smallest at 32-bit or 64-bit
template <bool reversed_order=false, bool scaling=SCALING, elementwise_func_t elementwise_func=DEFAULT_ELEMENTWISE,
          uint absmax_factor=ABSMAX_FACTOR, typename T1, typename T2>
__global__ void copy_advanced_kernel3(T1 *copy, const T2 *input, size_t N, const float* __restrict__ scale_pointer=d_scaling_factor, unsigned int* absmax_output=d_absmax_estimate, const void** meta=NULL) {
    // Optional fused absmax calculation
    uint absmax_uint = 0;
    // Optionally process in reverse order to maximise L2 cache hits across kernels for large tensors
    size_t adjusted_blockidx_x = reversed_order ? (gridDim.x - blockIdx.x - 1) : blockIdx.x;
    // Use the *smallest* of the two vector sizes in terms of elements (both are 128-bit if fully used)
    constexpr size_t vec_size = 16 / ((sizeof(T1) < sizeof(T2)) ? sizeof(T2) : sizeof(T1));
    size_t n = (adjusted_blockidx_x * blockDim.x + threadIdx.x) * vec_size;
    if (n >= N) { return; } // out of bounds check (todo - is this always OK when calculating absmax?)

    // note: if sizeof(T1) < sizeof(T2), compiler will skip unused elements of load128
    // so it may turn out to be a ldg.32 or ldg.64
    Packed128<T2> inp128;
    Packed128<T1> out128;
    inp128 = load128cs<T2>(input + n);
    float scale_factor = scaling ? *scale_pointer : 1.0f;
    for (int k = 0; k < vec_size; k++) {
        float out_float = elementwise_func((float)inp128[k], meta, n+k, 0, N, 1);
        out128[k] = (T1)(out_float * scale_factor);
        update_local_absmax(absmax_uint, out_float, absmax_factor); // optional absmax
    }
    // if sizeof(T2) < sizeof(T1), this will use stg.32 or stg.64 instead of stg.128
    store_same_length<T2,T1>(copy + n, out128);

    // update absmax if required
    if constexpr (absmax_factor != 0) {
        update_global_absmax(absmax_output, absmax_uint);
    }
}

// ----------------------------------------------------------------------------
// GPU kernels for transpose

// naive transpose kernel without shared memory or 128-bit load/store
template <bool scaling=SCALING, bool enable_copy=TRANSPOSE_AND_COPY, typename T1, typename T2>
__global__ void transpose_naive_kernel(T1 *transposed, T1* copy, const T2 *input, size_t width, size_t height,
                                       const float* __restrict__ scale_pointer=d_scaling_factor) {
    float scale_factor = scaling ? *scale_pointer : 1.0f;
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        T2 in = input[x + y * width];
        T1 out = scaling ? (T1)((float)in * scale_factor) : (T1)in;

        transposed[y + x*height] = out;
        if constexpr (enable_copy) {
            copy[x + y*width] = out;
        }
    }
}

// optimized transpose kernel with shared memory but *without* 128-bit load/store
// originally based on: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/transpose/transpose.cu
// also see this blog article: https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
// note that neither of these sources consider less than 32-bit data formats (and associated bank conflicts)
template<size_t BLOCK_ROWS=8UL, size_t TILE_DIM=DEFAULT_TILE, bool scaling=SCALING, bool enable_copy=TRANSPOSE_AND_COPY,
         elementwise_func_t elementwise_func=DEFAULT_ELEMENTWISE, typename T1, typename T2>
__global__ void transpose_kernel1(T1 *transposed, T1 *copy, const T2 *input,
                                  const float* __restrict__ scale_pointer=d_scaling_factor, const void** meta=NULL)
{
    __shared__ T1 tile[TILE_DIM][TILE_DIM+1]; // +1 for bank conflict avoidance
    int w = gridDim.x * TILE_DIM;
    int h = gridDim.y * TILE_DIM;

    float scale_factor = scaling ? *scale_pointer : 1.0f;
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        T2 in = input[x + (y+j)*w];
        float post_elementwise = elementwise_func((float)in, meta, x, y+j, w, h);
        T1 out = scaling ? (T1)(post_elementwise * scale_factor) : (T1)post_elementwise;

        tile[threadIdx.y+j][threadIdx.x] = out;
        if constexpr (enable_copy) {
            copy[x + (y+j)*w] = out; // separate copy with format conversion (on top of the transpose)
        }
    }
    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        // avoiding bank conflicts for 32-bit data types thanks to +1 above
        // (also seems to help sub-32-bit but less so, HW behaviour unclear)
        transposed[x + (y+j)*h] = tile[threadIdx.x][threadIdx.y + j];
    }
}

// more optimized transpose kernel using 128-bit load/store and shared memory
// only slightly faster by default, but much faster with TRANSPOSE_AND_COPY as sub-32-bit store in kernel1 is inefficient
template<size_t BLOCK_ROWS=8UL, size_t TILE_DIM=DEFAULT_TILE, bool scaling=SCALING, bool enable_copy=TRANSPOSE_AND_COPY,
         uint absmax_factor=ABSMAX_FACTOR, elementwise_func_t elementwise_func=DEFAULT_ELEMENTWISE, typename T1, typename T2>
__global__ void transpose_kernel2(T1* __restrict__ transposed, T1* __restrict__ copy, const T2* __restrict__ input,
                                  const float* __restrict__ scale_pointer=d_scaling_factor, unsigned int* absmax_output=d_absmax_estimate, const void** meta=NULL)
{
    // no +1 for bank conflict avoidance because:
    // 1) 128-bit shared memory stores need to be aligned to 128-bit boundaries
    // 2) it doesn't help as much with sub-32-bit data types
    __shared__ T1 tile[TILE_DIM][TILE_DIM];
    int w  = gridDim.x * TILE_DIM;
    int h = gridDim.y * TILE_DIM;

    constexpr size_t T1_elements = 16 / sizeof(T1);
    constexpr size_t T2_elements = 16 / sizeof(T2);
    constexpr size_t copy_vectors = (sizeof(T1) >= sizeof(T2)) ? (sizeof(T1) / sizeof(T2)) : 1;

    float scale_factor = scaling ? *scale_pointer : 1.0f;
    int x = blockIdx.x * TILE_DIM + (threadIdx.x * T2_elements);
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    uint absmax_uint = 0;

    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        Packed128<T2> in128 = load128cs<T2>(input + x + (y+j)*w);
        Packed128<T1> copy128[copy_vectors];
        for (int k = 0; k < in128.size; k++) {
            T2 in = in128[k];
            float out_float = elementwise_func((float)in, meta, x+k, y+j, w, h);

            T1 out = (T1)(out_float * scale_factor);
            copy128[k/T1_elements][k%T1_elements] = out; // optimised away by compiler if unused
            update_local_absmax(absmax_uint, out_float, absmax_factor); // optional absmax
        }

        for (int o = 0; o < copy_vectors; o++) {
            if constexpr (enable_copy) {
                store_same_length<T2,T1>(copy + x + (y+j)*w + o*T1_elements, copy128[o]);
            }
            size_t tile_offset = (threadIdx.x * T2_elements) + (threadIdx.y+j)*TILE_DIM + o*T1_elements;
            store_same_length<T2,T1>(&tile[0][0] + tile_offset, copy128[o]);
        }
    }

    if constexpr (absmax_factor != 0) {
        update_global_absmax<true>(absmax_output, absmax_uint);
    } else {
        __syncthreads();
    }

    // reduce the number of threads for the write if T1_elements > T2_elements
    // we want to keep all 32 threads in a warp active, so we try to eliminate in y dimension first
    // so we create fake/adjusted tid.x/tid.y where "extra" threadIdx.x adds to the effective tid.y
    constexpr size_t block_size_x = (TILE_DIM * sizeof(T2)) / 16;
    constexpr size_t block_size_y = BLOCK_ROWS;
    constexpr size_t desired_ratio = (sizeof(T2) >= sizeof(T1)) ? (sizeof(T2) / sizeof(T1)) : 1;
    constexpr size_t ratio = (desired_ratio <= block_size_y) ? desired_ratio : block_size_y;
    constexpr size_t block_size_x_div_r = block_size_x / ratio;
    constexpr size_t block_size_y_div_r = block_size_y / ratio;

    int adjusted_tid_x = threadIdx.x % block_size_x_div_r;
    int adjusted_tid_y = (threadIdx.y * ratio) + (threadIdx.x / block_size_x_div_r);
    if (threadIdx.y >= block_size_y_div_r) { return; }

    // if we cannot reduce block_size.y enough, also reduce x (hurting perf with partial warps)
    if (ratio != desired_ratio && adjusted_tid_x >= TILE_DIM / T1_elements) { return; }

    // x/y for final write to global memory
    x = blockIdx.y * TILE_DIM + adjusted_tid_x * T1_elements;
    y = blockIdx.x * TILE_DIM + adjusted_tid_y;

    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        // we need more instructions for the write than the read if T2_elements > T1_elements
        #pragma unroll
        for (int o = 0; o < copy_vectors; o++) {
            Packed128<T1> out128;
            #pragma unroll
            for (int k = 0; k < out128.size; k++) {
                // these are tiny 8-bit loads with loads of bank conflicts for FP8
                // extremely hard to avoid and not a bottleneck when everything else is well optimised
                out128[k] = tile[k + (adjusted_tid_x + o * blockDim.x) * out128.size][adjusted_tid_y + j];
            }
            store128<T1>(transposed + x + (o * blockDim.x * out128.size) + (y+j)*h, out128);
        }
    }
}

// best I could come up with (without using TMA) - no bank conflicts, but 64B reads/writes not ideal
// Z_DIM=2 improves perf by ~2% partly by improving L2 hit rates for the writes as far as I can tell
template<size_t BLOCK_ROWS=8UL, size_t TILE_DIM=DEFAULT_TILE, bool scaling=SCALING, bool enable_copy=TRANSPOSE_AND_COPY,
         uint absmax_factor=ABSMAX_FACTOR, elementwise_func_t elementwise_func=DEFAULT_ELEMENTWISE, int Z_DIM=DEFAULT_Z_DIM,
         typename T1, typename T2>
__global__ void transpose_kernel3(T1* __restrict__ transposed, T1* __restrict__ copy, const T2* __restrict__ input,
                                  const float* __restrict__ scale_pointer=d_scaling_factor, unsigned int* absmax_output=d_absmax_estimate,
                                  const void** meta=NULL, int height=HEIGHT)
{
    constexpr int in_parallel = 4/sizeof(T1);

    constexpr size_t TILE_DIM_PADDED = (TILE_DIM * 33) / 32;
    __shared__ T1 tile[Z_DIM][TILE_DIM][TILE_DIM_PADDED];
    int w  = gridDim.x * TILE_DIM;

    constexpr size_t T1_elements = 16 / sizeof(T1);
    constexpr size_t T2_elements = 16 / sizeof(T2);
    constexpr size_t copy_vectors = (sizeof(T1) >= sizeof(T2)) ? (sizeof(T1) / sizeof(T2)) : 1;

    float scale_factor = scaling ? *scale_pointer : 1.0f;
    int x = blockIdx.x * TILE_DIM + (threadIdx.x * T2_elements);
    int y = blockIdx.y * TILE_DIM * Z_DIM + threadIdx.z * TILE_DIM + threadIdx.y;

    uint absmax_uint = 0;
    if (y < height) {
        #pragma unroll
        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            Packed128<T1> copy128[copy_vectors];

            int4 payload;
            const int4* address = reinterpret_cast<const int4*>(input + x + (y+j)*w);
            asm volatile("ld.global.L2::128B.v4.s32 {%0, %1, %2, %3}, [%4];"
                        : "=r"(payload.x), "=r"(payload.y), "=r"(payload.z), "=r"(payload.w)
                        : "l"(address));
            Packed128<T2> in128(payload);

            #pragma unroll
            for (int k = 0; k < in128.size; k++) {
                T2 in = in128[k];
                float out_float = elementwise_func((float)in, meta, x+k, y+j, w, height);

                T1 out = (T1)(out_float * scale_factor);
                copy128[k/T1_elements][k%T1_elements] = out; // optimised away by compiler if unused
                update_local_absmax(absmax_uint, out_float, absmax_factor); // optional absmax
            }

            #pragma unroll
            for (int o = 0; o < copy_vectors; o++) {
                if constexpr (enable_copy) {
                    store_same_length<T2,T1>(copy + x + (y+j)*w + o*T1_elements, copy128[o]);
                }

                size_t offset_x = (threadIdx.x * T2_elements) + (o * T1_elements);
                size_t offset_y = (threadIdx.y + j) * TILE_DIM;
                offset_y += (offset_y / (128/sizeof(T1))) * in_parallel;

                int* one_bank = reinterpret_cast<int*>(&tile[threadIdx.z][0][0] + offset_x + offset_y);
                #pragma unroll
                for (int k = 0; k < 4; k++) {
                    one_bank[k] = *(int*)(&copy128[o][k*4/sizeof(T1)]);
                }
            }
        }
    }

    if constexpr (absmax_factor != 0) {
        update_global_absmax<true, true>(absmax_output, absmax_uint);
    } else {
        __syncthreads();
    }
    if (y >= height) { return; }

    // reduce the number of threads for the write if T1_elements > T2_elements
    // we want to keep all 32 threads in a warp active, so we try to eliminate in y dimension first
    // so we create fake/adjusted tid.x/tid.y where "extra" threadIdx.x adds to the effective tid.y
    constexpr size_t block_size_x = (TILE_DIM * sizeof(T2)) / 16;
    constexpr size_t block_size_y = BLOCK_ROWS;
    constexpr size_t desired_ratio = (sizeof(T2) >= sizeof(T1)) ? (sizeof(T2) / sizeof(T1)) : 1;
    constexpr size_t ratio = (desired_ratio <= block_size_y) ? desired_ratio : block_size_y;
    constexpr size_t block_size_x_div_r = block_size_x / ratio;
    constexpr size_t block_size_y_div_r = block_size_y / ratio;

    int adjusted_tid_x = threadIdx.x % block_size_x_div_r;
    int adjusted_tid_y = (threadIdx.y * ratio) + (threadIdx.x / block_size_x_div_r);
    if (threadIdx.y >= block_size_y_div_r) { return; }

    // if we cannot reduce block_size.y enough, also reduce x (hurting perf with partial warps)
    if (ratio != desired_ratio && adjusted_tid_x >= TILE_DIM / T1_elements) { return; }

    // x/y for final write to global memory
    x = blockIdx.y * TILE_DIM * Z_DIM + threadIdx.z * TILE_DIM + adjusted_tid_x * T1_elements;
    y = blockIdx.x * TILE_DIM + (adjusted_tid_y*in_parallel);

    #pragma unroll
    for (int j = 0; j < TILE_DIM / in_parallel; j += BLOCK_ROWS) {
        if ((j+adjusted_tid_y) * in_parallel >= TILE_DIM) { return; }

        // we need more instructions for the write than the read if T2_elements > T1_elements
        #pragma unroll
        for (int o = 0; o < copy_vectors; o++) {
            Packed128<T1> out128[in_parallel];
            #pragma unroll
            for (int k = 0; k < Packed128<T1>::size; k++) {
                int offset_x = (adjusted_tid_y + j) * in_parallel;
                int offset_y = ((adjusted_tid_x + o * blockDim.x) * Packed128<T1>::size + k) * TILE_DIM;
                offset_y += (offset_y / (128/sizeof(T1))) * in_parallel;

                int in32 = *(int*)(&tile[threadIdx.z][0][0] + offset_x + offset_y);
                for (int p = 0; p < in_parallel; p++) {
                    out128[p][k] = ((T1*)&in32)[p];
                }
            }
            #pragma unroll
            for (int p = 0; p < in_parallel; p++) {
                store128<T1>(transposed + x + (o * blockDim.x * Packed128<T1>::size) + (y+p + j * in_parallel) * height, out128[p]);
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels for absmax

// kernel to calculate absmax of the input tensor
template <typename T=IN_TYPE>
__global__ void get_absmax_kernel(const T* inp, unsigned int* absmax_output, size_t N, uint absmax_factor=1) {
    uint absmax_uint = 0;

    size_t idx = ((blockIdx.x * blockDim.x * ABSMAX_ITERATIONS_PER_THREAD) + threadIdx.x) * Packed128<T>::size;
    if (idx < N) {
        for (int i = 0; i < ABSMAX_ITERATIONS_PER_THREAD; i++) {
            Packed128<T> packed_inp = load128(inp + idx);
            for(int k = 0; k < packed_inp.size; ++k) {
                update_local_absmax<true>(absmax_uint, packed_inp[k], absmax_factor); // optional absmax
            }
            idx += blockDim.x * packed_inp.size;
        }
    }
    update_global_absmax<false>(absmax_output, absmax_uint);
}

template <typename T=IN_TYPE>
__global__ void get_absmax_persistent_kernel(const T* inp, unsigned int* absmax_output, size_t N, uint absmax_factor=1) {
    int elements_per_block_iteration = blockDim.x * ABSMAX_ITERATIONS_PER_THREAD * Packed128<T>::size;
    int iterations_per_block = (int)ceil_div(N, (size_t)(gridDim.x * elements_per_block_iteration));
    size_t start_idx = blockIdx.x * elements_per_block_iteration * iterations_per_block;
    uint absmax_uint = 0;

    for (size_t i = 0; i < iterations_per_block; i++) {
        size_t idx = start_idx + (i * elements_per_block_iteration) + (threadIdx.x * Packed128<T>::size);
        if (idx < N) {
            for (int o = 0; o < ABSMAX_ITERATIONS_PER_THREAD; o++) {
                Packed128<T> packed_inp = load128(inp + idx);
                for(int k = 0; k < packed_inp.size; ++k) {
                    update_local_absmax<true>(absmax_uint, packed_inp[k], absmax_factor);
                }
                idx += blockDim.x * packed_inp.size;
            }
        }
    }
    update_global_absmax<false>(absmax_output, absmax_uint);
}

template <int NUM_WARPS=32, typename T=IN_TYPE, typename TOut=OUT_TYPE>
__global__ void __launch_bounds__(1024, 2) fused_absmax_scale_persistent(TOut* __restrict__ out, unsigned int* __restrict__ absmax_scaling, unsigned int* __restrict__ absmax_actual, unsigned int* __restrict__ absmax_counter,
                                                                         const T* inp, size_t N, uint absmax_factor=1) {
    // todo - move these calculations to CPU?
    int elements_per_block_iteration = blockDim.x * ABSMAX_ITERATIONS_PER_THREAD * Packed128<T>::size;
    int iterations_per_block = (int)ceil_div(N, (size_t)(gridDim.x * elements_per_block_iteration));
    size_t start_idx = blockIdx.x * elements_per_block_iteration * iterations_per_block;
    unsigned int absmax_uint = 0;

    // iterations to get to 40MiB read
    int bytes_per_iteration = gridDim.x * elements_per_block_iteration * sizeof(T);
    int iterations_for_first_phase = min(iterations_per_block, 1 + (FUSED_ABSMAX_FIRST_PHASE_BYTES / bytes_per_iteration));

    Packed128<T> packed_inp[ABSMAX_ITERATIONS_PER_THREAD];
    for (int i = iterations_for_first_phase-1; i >= 0; i--) {
        size_t idx = start_idx + (i * elements_per_block_iteration) + (threadIdx.x * Packed128<T>::size);
        if (idx < N) {
            for (int o = 0; o < ABSMAX_ITERATIONS_PER_THREAD; o++) {
                // i=0 is cached in registers, so use load128cs to not keep in L2
                packed_inp[o] = (i == 0) ? load128cs(inp + idx) : load128(inp + idx);
                for(int k = 0; k < packed_inp[o].size; ++k) {
                    update_local_absmax<true>(absmax_uint, packed_inp[o][k], absmax_factor);
                }
                idx += blockDim.x * packed_inp[o].size;
            }
        }
    }
    update_global_absmax<false>(absmax_scaling, absmax_uint);
    __threadfence(); // make sure the atomicInc always happens after the atomicMax
    if (threadIdx.x == 0) {
        atomicInc(absmax_counter, gridDim.x-1); // increment the number of blocks done with phase 1
    }
    __syncthreads();

    // Prefetch the very start of the next iteration so the DRAM controller has something to do
    size_t idx_prefetch = start_idx + (iterations_for_first_phase * elements_per_block_iteration) + (threadIdx.x * Packed128<T>::size);
    if (idx_prefetch < N) {
        asm volatile("prefetch.global.L1 [%0];" :: "l"(inp + idx_prefetch));
    }

    // Wait until all blocks have incremented the counter indicating they are done with phase 1
    __shared__ unsigned int absmax_shared;
    if (threadIdx.x == 0) {
        // volatile read of absmax_counter: wait until it is reset to 0 (because val = gridDim.x-1)
        bool done = (__ldcg(absmax_counter) == 0);
        while (!done) {
            __nanosleep(100); // sleep for 100 nanoseconds, i.e. 200 cycles at 2GHz, then retry
            done = (__ldcg(absmax_counter) == 0);
        }
        absmax_shared = __ldcg(absmax_scaling);
    }
    __syncthreads();
    unsigned int absmax_uint_used = absmax_shared;
    float estimated_absmax = __uint_as_float(absmax_uint_used);

    // Prefetch the 2nd part of the next iteration so the DRAM controller has something to do
    if (idx_prefetch < N && ABSMAX_ITERATIONS_PER_THREAD >= 2) {
        asm volatile("prefetch.global.L1 [%0];" :: "l"(inp + idx_prefetch));
    }

    // Now we can do the actual scaling for the 1st iteration which we cached in the packed_inp registers
    int i = 0;
    size_t idx = start_idx + (i * elements_per_block_iteration) + (threadIdx.x * Packed128<T>::size);
    if (idx < N) {
        for (int o = 0; o < ABSMAX_ITERATIONS_PER_THREAD; o++) {
            Packed128<TOut> packed_out;
            for(int k = 0; k < packed_inp[o].size; ++k) {
                packed_out[k] = (TOut)((float)packed_inp[o][k] / estimated_absmax);
            }
            store_same_length<T,TOut>(out + idx, packed_out);
            idx += blockDim.x * Packed128<T>::size;
        }
    }

    int last_scaled = 0;
    bool scale_output = true;

    // We do the scaling for everything else, while keeping track of the absmax
    // if the absmax no longer matches, stop copying and only calculate absmax, then copy everything at the end
    for (int i = 1; i < iterations_per_block; i++) {
        size_t idx = start_idx + (i * elements_per_block_iteration) + (threadIdx.x * Packed128<T>::size);
        if (idx < N) {
            for (int o = 0; o < ABSMAX_ITERATIONS_PER_THREAD; o++) {
                Packed128<TOut> packed_out;
                Packed128<T> packed_inp = load128cs(inp + idx); // last read, do not cache in either L1 or L2
                for(int k = 0; k < packed_inp.size; ++k) {
                    update_local_absmax<true>(absmax_uint, packed_inp[k], absmax_factor);
                    packed_out[k] = (TOut)((float)packed_inp[k] / estimated_absmax);
                }
                if (scale_output) {
                    store_same_length<T,TOut>(out + idx, packed_out);
                }
                idx += blockDim.x * Packed128<T>::size;
            }
        }
        #if FUSED_RESCALE_IN_PLACE == true
        // per-warp "stop scaling if we have ever seen a value too big" (effectively discarding the iteration above)
        // doing it per-warp rather than per-block reduces sync overhead (and we end up using in-place rescaling a lot)
        if (scale_output) {
            asm volatile("redux.sync.max.u32 %0, %0, 0xff;" : "+r"(absmax_uint));
            float absmax_this_iteration = __uint_as_float(absmax_uint);
            if (absmax_this_iteration > estimated_absmax) {
                scale_output = false;
            } else {
                last_scaled = i;
            }
        }
        #endif
    }
    (void)last_scaled; (void)scale_output; // avoid compiler warnings if not used

    // Now update the global max and wait until all other blocks are done to recheck the absmax
    update_global_absmax<false>(absmax_actual, absmax_uint);

    // todo - this is a WIP path that rescales the tensor in-place if it was already scaled to FP8,
    // and scales from BF16 input for every part that has not been scaled (since we knew the scale was wrong)
    // NOT PROPERLY TESTED YET - DO NOT TRUST IT TO WORK CORRECTLY!
    #if FUSED_RESCALE_IN_PLACE == true
    if (warp_id == 0) {
        if (threadIdx.x == 0) {
            unsigned int old = atomicInc(absmax_counter, gridDim.x-1);
            bool done = (old == gridDim.x-1);
            while (!done) {
                __nanosleep(100); // sleep for 100 nanoseconds, i.e. 200 cycles at 2GHz, then retry
                done = (__ldcg(absmax_counter) == 0);
            }
            tmp[0] = __ldcg(absmax_actual);
        }
    }
    __syncthreads();

    absmax_uint = tmp[0];
    float final_absmax = __uint_as_float(absmax_uint);

    if (final_absmax <= estimated_absmax) { return; } // no rescaling needed, we're done!
    // We need to rescale everything we did output! :(
    // We scale the FP8 tensor in-place by a power of 2 so it only affects the exponent bits
    // (except for subnormals and special numbers, but because this is a persistent kernel, it's still deterministic)

    // If we did this naively, we'd waste half the warps in a block when size(T) > sizeof(TOut) (e.g. BF16->FP8)
    // instead we make different warps do different loop iterations to maintain 100% occupancy
    constexpr int size_ratio = sizeof(T) / sizeof(TOut);
    int start_o = 0;
    int step_o = 1;
    int tid_x = threadIdx.x;
    if (size_ratio > 1) {
        constexpr int threads_per_o = (NUM_WARPS*32) / size_ratio;
        tid_x = threadIdx.x % threads_per_o;
        start_o = threadIdx.x / threads_per_o;
        step_o = size_ratio;
    }

    float ratio = final_absmax / estimated_absmax;
    float ratio_power_of_2 = exp2f(ceil(__log2f(ratio)));
    float rescale = 1.0f / ratio_power_of_2;

    // rescale the the incorrectly scaled FP8 outputs with the new scale
    // iterate in reverse so we are more likely to hit in the cache
    for (int i = last_scaled; i >= 0; i--) {
        size_t idx = start_idx + (i * elements_per_block_iteration) + (tid_x * Packed128<TOut>::size);
        if (idx < N) {
            for (int o = start_o; o < ABSMAX_ITERATIONS_PER_THREAD; o += step_o) {
                Packed128<TOut> packed_in_out = load128(out + idx);
                for(int k = 0; k < Packed128<TOut>::size; ++k) {
                    packed_in_out[k] = (TOut)((float)packed_in_out[k] * rescale);
                }
                store128cs<TOut>(out + idx, packed_in_out);
            }
        }
    }

    // Go through all the iterations we did *not* scale on this warp because we knew the scale was wrong
    for (int i = last_scaled+1; i < iterations_per_block; i++) {
        size_t idx = start_idx + (i * elements_per_block_iteration) + (tid_x * Packed128<T>::size);
        if (idx < N) {
            for (int o = start_o; o < ABSMAX_ITERATIONS_PER_THREAD; o += step_o) {
                Packed128<TOut> packed_out;
                Packed128<T> packed_inp = load128cs(inp + idx); // last read, do not cache in either L1 or L2
                for(int k = 0; k < packed_inp.size; ++k) {
                    packed_out[k] = (TOut)((float)packed_inp[k] / final_absmax);
                }
                store_same_length<T,TOut>(out + idx, packed_out);
                idx += blockDim.x * packed_inp.size;
            }
        }
    }

    // Update absmax_scaling so other kernels know what to descale by
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        float rescaled_absmax = estimated_absmax * ratio_power_of_2;
        *absmax_scaling = __float_as_uint(rescaled_absmax);
    }
    #endif
}

// ----------------------------------------------------------------------------
// kernel launchers

template <typename T1, typename T2>
void copy_naive_0(T1 *copy, const T2 *input, size_t width, size_t height, const size_t block_size) {
    size_t N = width * height;
    const dim3 grid_size(ceil_div(N, block_size));
    copy_naive_kernel0<<<grid_size, dim3(block_size)>>>(copy, input, N);
}

template <typename T1, typename T2>
void copy_fast_1(T1 *copy, const T2 *input, size_t width, size_t height, const size_t block_size) {
    size_t N = width * height;
    size_t fewest_elements = min(Packed128<T1>::size, Packed128<T2>::size);
    const dim3 grid_size(ceil_div(N, block_size * fewest_elements));
    copy_fast_kernel1<<<grid_size, dim3(block_size)>>>(copy, input, N);
}

template <typename T1, typename T2>
void copy_advanced_2(T1 *copy, const T2 *input, size_t width, size_t height, const size_t block_size) {
    size_t N = width * height;
    const dim3 grid_size(ceil_div(N, block_size * 16 / sizeof(T1)));
    copy_advanced_kernel2<<<grid_size, dim3(block_size)>>>(copy, input, N);
}

template <typename T1, typename T2>
void copy_advanced_3(T1 *copy, const T2 *input, size_t width, size_t height, const size_t block_size) {
    size_t N = width * height;
    size_t fewest_elements = min(Packed128<T1>::size, Packed128<T2>::size);
    const dim3 grid_size(ceil_div(N, block_size * fewest_elements));
    copy_advanced_kernel3<<<grid_size, dim3(block_size)>>>(copy, input, N);
}

template <typename T1, typename T2>
void transpose_naive(T1 *transposed, const T2 *input, size_t width, size_t height, const size_t block_size, T1 *copy=NULL) {
    // actual block size is sqrt(block_size) rounded to next power of 2 (so 128 is really 256 unfortunately...)
    size_t actual_block_size = 1 << (int)ceil(log2(sqrt(block_size)));
    const dim3 grid_size(ceil_div(width, block_size), ceil_div(height, block_size));
    transpose_naive_kernel<<<grid_size, dim3(actual_block_size)>>>(transposed, copy, input, width, height);
}

template <typename T1, typename T2>
void transpose1(T1 *transposed, const T2 *input, size_t width, size_t height, const size_t block_size, T1 *copy=NULL) {
    dim3 grid_size(width / DEFAULT_TILE, height / DEFAULT_TILE);
    dim3 block_size_(DEFAULT_TILE, max(1UL, block_size / DEFAULT_TILE)); // always >=1, so might not respect block size for large tiles

    switch (block_size_.y) {
        case 32: transpose_kernel1<32, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 16: transpose_kernel1<16, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 8: transpose_kernel1<8, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 4: transpose_kernel1<4, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 2: transpose_kernel1<2, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 1: transpose_kernel1<1, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        default: printf("Invalid block size: %d\n", block_size_.y); exit(1);
    }
}

template <typename T1, typename T2>
void transpose2(T1 *transposed, const T2 *input, size_t width, size_t height, const size_t block_size, T1 *copy=NULL) {
    size_t block_size_x = (DEFAULT_TILE * sizeof(T2)) / 16;
    size_t block_size_y = min(DEFAULT_TILE, block_size / block_size_x);
    dim3 grid_size(width / DEFAULT_TILE, height / DEFAULT_TILE);
    dim3 block_size_(block_size_x, block_size_y);

    switch (block_size_y) {
        case 128: transpose_kernel2<128, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 64: transpose_kernel2<64, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 32: transpose_kernel2<32, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 16: transpose_kernel2<16, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 8: transpose_kernel2<8, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 4: transpose_kernel2<4, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 2: transpose_kernel2<2, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 1: transpose_kernel2<1, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        default: printf("Invalid block size: %lu\n", block_size_y); exit(1);
    }
}

template <typename T1, typename T2>
void transpose3(T1 *transposed, const T2 *input, size_t width, size_t height, const size_t block_size, T1 *copy=NULL) {
    size_t block_size_x = (DEFAULT_TILE * sizeof(T2)) / 16;
    size_t block_size_y = min(DEFAULT_TILE, block_size / block_size_x);
    dim3 grid_size(width / (DEFAULT_TILE), height / (DEFAULT_TILE*DEFAULT_Z_DIM), 1);
    dim3 block_size_(block_size_x, block_size_y, DEFAULT_Z_DIM);

    switch (block_size_y) {
        case 128: transpose_kernel3<128, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 64: transpose_kernel3<64, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 32: transpose_kernel3<32, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 16: transpose_kernel3<16, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 8: transpose_kernel3<8, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 4: transpose_kernel3<4, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 2: transpose_kernel3<2, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        case 1: transpose_kernel3<1, DEFAULT_TILE, SCALING, TRANSPOSE_AND_COPY><<<grid_size, block_size_>>>(transposed, copy, input); break;
        default: printf("Invalid block size: %lu\n", block_size_y); exit(1);
    }
}

template <typename T>
void get_absmax(const T* input, size_t N, const size_t block_size, bool memset=true, unsigned int* absmax_output=d_absmax_estimate, uint absmax_factor=ABSMAX_FACTOR) {
    size_t grid_size = ceil_div(N, block_size * Packed128<T>::size * ABSMAX_ITERATIONS_PER_THREAD);
    absmax_factor = absmax_factor ? absmax_factor : (float)DEFAULT_ABSMAX_FACTOR;
    //assert((N % (Packed128<T>::size * ABSMAX_ITERATIONS_PER_THREAD)) == 0);

    if (memset) {
        hipMemset(absmax_output, 0, sizeof(unsigned int));
    }
    get_absmax_kernel<<<grid_size, block_size>>>(input, absmax_output, N, absmax_factor);
    cudaCheck(hipGetLastError());
}

template <bool reversed_copy=false, typename T1, typename T2>
void absmax_and_copy(T1* copy, const T2* input, size_t N, const size_t block_size, bool memset=true, unsigned int* absmax_output=d_absmax_estimate, uint absmax_factor=ABSMAX_FACTOR) {
    get_absmax(input, N, block_size, false, absmax_output, absmax_factor);

    size_t fewest_elements = min(Packed128<T1>::size, Packed128<T2>::size);
    const dim3 grid_size_copy(ceil_div(N, block_size * fewest_elements));

    copy_advanced_kernel3<reversed_copy><<<grid_size_copy, dim3(block_size)>>>(copy, input, N);
    cudaCheck(hipGetLastError());
}

template <typename T>
void get_absmax_persistent(const T* input, size_t N, const size_t block_size, bool memset=true, unsigned int* absmax_output=d_absmax_estimate, uint absmax_factor=ABSMAX_FACTOR) {
    size_t grid_size = cuda_num_SMs * min(32, (int)(cuda_threads_per_SM / block_size)); // maximum of 32 blocks in flight
    absmax_factor = absmax_factor ? absmax_factor : DEFAULT_ABSMAX_FACTOR;
    //assert((N % (Packed128<T>::size * ABSMAX_ITERATIONS_PER_THREAD)) == 0);

    if (memset) {
        hipMemset(absmax_output, 0, sizeof(unsigned int));
    }
    get_absmax_persistent_kernel<<<grid_size, block_size>>>(input, absmax_output, N, absmax_factor);
    cudaCheck(hipGetLastError());
}

template <typename T1, typename T2>
void fused_absmax_scale_persistent(T1* out, const T2* input, size_t N, const size_t block_size, bool memset=true, uint absmax_factor=ABSMAX_FACTOR) {
    size_t grid_size = cuda_num_SMs * min(32, (int)(cuda_threads_per_SM / block_size)); // maximum of 32 blocks in flight
    absmax_factor = absmax_factor ? absmax_factor : DEFAULT_ABSMAX_FACTOR;
    //assert((N % (Packed128<T1>::size * ABSMAX_ITERATIONS_PER_THREAD)) == 0);

    if (memset) {
        hipMemset(d_absmax_estimate, 0, sizeof(unsigned int));
        hipMemset(d_absmax_counter, 0, sizeof(unsigned int));
        hipMemset(d_absmax_actual, 0, sizeof(unsigned int));
    }
    cudaCheck(hipGetLastError());

    // todo - ideally this should use cooperative thread launches so that the CUDA API itself guarantees all blocks can execute simultaneously
    switch (block_size) {
        case 32: fused_absmax_scale_persistent<1><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 64: fused_absmax_scale_persistent<2><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 128: fused_absmax_scale_persistent<4><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 256: fused_absmax_scale_persistent<8><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 512: fused_absmax_scale_persistent<16><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 768: fused_absmax_scale_persistent<24><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        case 1024: fused_absmax_scale_persistent<32><<<grid_size, block_size>>>(out, d_absmax_estimate, d_absmax_actual, d_absmax_counter, input, N, absmax_factor); break;
        default: printf("Invalid block size: %lu\n", block_size); exit(1);
    }
    cudaCheck(hipGetLastError());
}


// kernel version dispatch
template <typename T1, typename T2>
void run_advanced(int kernel_num,
                  T1 *transposed, T1 *copy, const T2 *input,
                  size_t width, size_t height, size_t block_size) {
    switch (kernel_num) {
        case 0:
            copy_naive_0(copy, input, width, height, block_size);
            break;
        case 1:
            copy_fast_1(copy, input, width, height, block_size);
            break;
        case 2:
            copy_advanced_2(copy, input, width, height, block_size);
            break;
        case 3:
            // our best copy with the most features
            copy_advanced_3(copy, input, width, height, block_size);
            break;
        case 10:
            transpose_naive(transposed, input, width, height, block_size, copy);
            break;
        case 11:
            transpose1(transposed, input, width, height, block_size, copy);
            break;
        case 12:
            // our best transpose with the most features
            transpose2(transposed, input, width, height, block_size, copy);
            break;
        case 13:
            transpose3(transposed, input, width, height, block_size, copy);
            break;
        case 20:
            get_absmax(input, width * height, block_size, true);
            break;
        case 21:
            // no memset (negligible difference except for tiny tensors)
            get_absmax(input, width * height, block_size, false);
            break;
        case 22:
            get_absmax_persistent(input, width * height, block_size, true);
            break;
        case 25:
            absmax_and_copy(copy, input, width * height, block_size, true);
            break;
        case 26:
            // reversed copy which leads to some L2 cache hits on copy after absmax
            absmax_and_copy<true>(copy, input, width * height, block_size, true);
            break;
        case 30:
            fused_absmax_scale_persistent(copy, input, width * height, block_size, true);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------

int main(int argc, const char **argv) {
    setup_main();
    int W = WIDTH;
    int H = HEIGHT;

    // create host memory of random numbers (0 to 1 so there's no overflow with format conversion)
    OUT_TYPE* transposed = (OUT_TYPE*)malloc(W * H * sizeof(OUT_TYPE));
    OUT_TYPE* copy = (OUT_TYPE*)malloc(W * H * sizeof(OUT_TYPE));
    OUT_TYPE* out = (OUT_TYPE*)malloc(W * H * sizeof(OUT_TYPE));
    OUT_TYPE* copy_gelu = (OUT_TYPE*)malloc(W * H * sizeof(OUT_TYPE));
    OUT_TYPE* transposed_gelu = (OUT_TYPE*)malloc(W * H * sizeof(OUT_TYPE));
    float* input = make_random_float_01(W * H);

    // add an outlier towards the end to make the job of fused absmax really hard
    //input[(W/7) + ((H*4)/5)*W] = 435.0f;
    input[W-1 + (H-1)*W] = 1000.0f;

    // read kernel_num from command line
    int kernel_num = 12;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    transpose_cpu(transposed, transposed_gelu, copy, copy_gelu, input, W, H);

    // move to GPU
    IN_TYPE *d_input;
    OUT_TYPE *d_transposed, *d_copy;
    cudaCheck(hipMalloc(&d_transposed, W * H * sizeof(OUT_TYPE)));
    cudaCheck(hipMalloc(&d_copy, W * H * sizeof(OUT_TYPE)));
    cudaCheck(hipMalloc(&d_input, W * H * sizeof(IN_TYPE)));
    cudaCheck(memcpy_convert(d_input, input, W * H));

    float scaling_factor = SCALING_FACTOR;
    cudaCheck(hipMalloc(&d_scaling_factor, sizeof(float)));
    cudaCheck(hipMemcpy(d_scaling_factor, &scaling_factor, sizeof(float), hipMemcpyHostToDevice));

    cudaCheck(hipMalloc(&d_absmax_estimate, sizeof(unsigned int)));
    cudaCheck(hipMalloc(&d_absmax_counter, sizeof(unsigned int)));
    cudaCheck(hipMalloc(&d_absmax_actual, sizeof(unsigned int)));
    cudaCheck(hipMemset(d_absmax_estimate, 0, sizeof(unsigned int)));
    cudaCheck(hipMemset(d_absmax_counter, 0, sizeof(unsigned int)));
    cudaCheck(hipMemset(d_absmax_actual, 0, sizeof(unsigned int)));

    // time the kernel at different block sizes
    int block_sizes[] = {64, 128, 256, 512, 1024};

    // kernel 12 specifically does not support all block sizes, so act accordingly
    size_t num_block_sizes = sizeof(block_sizes) / sizeof(int);
    if (kernel_num == 12 || kernel_num == 13) {
        size_t block_size_x = (DEFAULT_TILE * sizeof(OUT_TYPE)) / 16;
        size_t block_size_y = min(DEFAULT_TILE, 1024 / block_size_x);
        size_t max_block_size = block_size_y * block_size_x;
        while (block_sizes[num_block_sizes - 1] > max_block_size && num_block_sizes >= 1) {
            num_block_sizes--;
        }
    }

    bool enable_gelu = ENABLE_GELU && kernel_num != 0 && kernel_num != 1 && kernel_num != 10;
    bool enable_absmax = CALCULATE_ABSMAX && kernel_num != 0 && kernel_num != 1 && kernel_num != 10 && kernel_num != 11;

    #if ENABLE_GELU == true
    if (!enable_gelu)
        printf("WARNING: This kernel does not support GELU calculation.\n");
    #endif
    #if CALCULATE_ABSMAX == true
    if (!enable_absmax)
        printf("WARNING: This kernel does not support absmax calculation.\n");
    #endif

    for (int j = 0; j < num_block_sizes; j++) {
        printf("Checking block size %d.\n", block_sizes[j]);
        run_advanced(kernel_num, d_transposed, d_copy, d_input, W, H, block_sizes[j]);

        if (kernel_num < FIRST_ABSMAX_ONLY_KERNEL) {
            // check copy tensor for copy kernels & for all others in +copy mode
            if (kernel_num < FIRST_TRANSPOSE_KERNEL || TRANSPOSE_AND_COPY == true) {
                if (enable_gelu) {
                    validate_result(d_copy, copy_gelu, "copy_gelu", W * H, (OUT_TYPE)1e-5f);
                } else {
                    validate_result(d_copy, copy, "copy", W * H, (OUT_TYPE)1e-5f);
                }
            }

            // check transposed tensor for transpose kernels
            if (kernel_num >= FIRST_TRANSPOSE_KERNEL) {
                if (enable_gelu) {
                    validate_result(d_transposed, transposed_gelu, "transposed_gelu", W * H, (OUT_TYPE)1e-5f);
                } else {
                    validate_result(d_transposed, transposed, "transposed", W * H, (OUT_TYPE)1e-5f);
                }
            }
        }

        // check absmax if it was calculated
        if (enable_absmax || kernel_num >= FIRST_ABSMAX_ONLY_KERNEL) {
            if (kernel_num != 30) { // don't check for the WIP fused absmax kernel yet
                validate_result((float*)d_absmax_estimate, (float*)&absmax_storage, "absmax", 1, 1e-5f);
            }
        }
        // todo - sanity checks specifically for kernel 30
    }
    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < num_block_sizes; j++) {
        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, run_advanced<OUT_TYPE, IN_TYPE>,
                                              kernel_num, d_transposed, d_copy, d_input,
                                              W, H, block_sizes[j]);

        // napkin math: estimate the memory bandwidth achieved
        size_t memory_ops = W * H * (sizeof(IN_TYPE) + sizeof(OUT_TYPE));
        #if TRANSPOSE_AND_COPY == true
        if (kernel_num >= FIRST_TRANSPOSE_KERNEL && kernel_num < FIRST_ABSMAX_ONLY_KERNEL) {
            memory_ops += W * H * sizeof(OUT_TYPE);
        }
        #endif
        if (kernel_num >= FIRST_ABSMAX_ONLY_KERNEL && kernel_num != 30) {
            if (kernel_num < 25) {
                memory_ops = 0; // 20/21/22 only do the absmax, no copy
            }
            memory_ops += W * H * sizeof(IN_TYPE); // read-only absmax kernel (+copy for 22/23)
        }
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;
        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_sizes[j], elapsed_time, memory_bandwidth);
    }

    free(out);
    free(copy);
    free(input);
    free(transposed);
    free(copy_gelu);
    free(transposed_gelu);
    cudaCheck(hipFree(d_input));
    cudaCheck(hipFree(d_copy));
    cudaCheck(hipFree(d_transposed));
    cudaCheck(hipFree(d_scaling_factor));
}