#include "hip/hip_runtime.h"
/*
Kernels for layernorm forward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt layernorm_forward.cu -o
layernorm_forward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops
over C
./layernorm_forward 1

version 2 parallelizes over all of B,T,C
./layernorm_forward 2

version 3 uses cooperative groups to parallelize over all of B,T,C
./layernorm_forward 3

version 4 uses a more clever way to estimate variance, var(x) = mean(x**2) -
mean(x)**2 (allowing us to do a single pass over x on load)
./layernorm_forward 4

verstion 5 allocates blocks per row instead of warps per row, same alg as 4
otherwise
./layernorm_forward 5
*/
#include "common.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdlib.h>

// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 layernorm forward pass
void layernorm_forward_cpu(float *out, float *mean, float *rstd,
                           const float *inp, const float *weight,
                           const float *bias, int B, int T, int C) {
  float eps = 1e-5f;
  for (int b = 0; b < B; b++) {
    for (int t = 0; t < T; t++) {
      // seek to the input position inp[b,t,:]
      const float *x = inp + b * T * C + t * C;
      // calculate the mean
      float m = 0.0f;
      for (int i = 0; i < C; i++) {
        m += x[i];
      }
      m = m / C;
      // calculate the variance (without any bias correction)
      float v = 0.0f;
      for (int i = 0; i < C; i++) {
        float xshift = x[i] - m;
        v += xshift * xshift;
      }
      v = v / C;
      // calculate the rstd
      float s = 1.0f / sqrtf(v + eps);
      // seek to the output position in out[b,t,:]
      float *out_bt = out + b * T * C + t * C;
      for (int i = 0; i < C; i++) {
        float n = (s * (x[i] - m));        // normalized output
        float o = n * weight[i] + bias[i]; // scale and shift it
        out_bt[i] = o;                     // write
      }
      // cache the mean and rstd for the backward pass later
      mean[b * T + t] = m;
      rstd[b * T + t] = s;
    }
  }
}

// --------------------------------------------------------------------------
// Memory Management

// allocate & set host & device memory, pinned memory allocation if necessary
// pinned memcpy is interleaved with kernel invocation (hipMemcpyAsync)
// in order to achieve copy compute overlap
void prepareMemory(float **out, float **mean, float **rstd, float **inp,
                   float **weight, float **bias, float **d_out, float **d_mean,
                   float **d_rstd, float **d_inp, float **d_weight,
                   float **d_bias, int B, int T, int C, bool pinned = false) {

  srand(0);

  cudaCheck(hipMalloc(d_out, B * T * C * sizeof(float)));
  cudaCheck(hipMalloc(d_mean, B * T * sizeof(float)));
  cudaCheck(hipMalloc(d_rstd, B * T * sizeof(float)));
  cudaCheck(hipMalloc(d_inp, B * T * C * sizeof(float)));
  cudaCheck(hipMalloc(d_weight, C * sizeof(float)));
  cudaCheck(hipMalloc(d_bias, C * sizeof(float)));

  if (pinned) {
    cudaCheck(
        hipHostAlloc(out, B * T * C * sizeof(float), hipHostMallocDefault));
    cudaCheck(hipHostAlloc(mean, B * T * sizeof(float), hipHostMallocDefault));
    cudaCheck(hipHostAlloc(rstd, B * T * sizeof(float), hipHostMallocDefault));

    *inp = make_random_float_pinned(B * T * C);
    *weight = make_random_float_pinned(C);
    *bias = make_random_float_pinned(C);
  } else {
    *out = (float *)malloc(B * T * C * sizeof(float));
    *mean = (float *)malloc(B * T * sizeof(float));
    *rstd = (float *)malloc(B * T * sizeof(float));

    *inp = make_random_float(B * T * C);
    *weight = make_random_float(C);
    *bias = make_random_float(C);

    cudaCheck(hipMemcpy(*d_inp, *inp, B * T * C * sizeof(float),
                         hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(*d_weight, *weight, C * sizeof(float),
                         hipMemcpyHostToDevice));
    cudaCheck(
        hipMemcpy(*d_bias, *bias, C * sizeof(float), hipMemcpyHostToDevice));
  }
}

void prepareCPUMemory(float **out, float **mean, float **rstd, float **inp,
                      float **weight, float **bias, int B, int T, int C) {

  srand(0);

  *out = (float *)malloc(B * T * C * sizeof(float));
  *mean = (float *)malloc(B * T * sizeof(float));
  *rstd = (float *)malloc(B * T * sizeof(float));
  *inp = make_random_float(B * T * C);
  *weight = make_random_float(C);
  *bias = make_random_float(C);
}

// free memory, pinned memory requires hipHostFree
void resetMemory(float **out, float **mean, float **rstd, float **inp,
                 float **weight, float **bias, float **d_out, float **d_mean,
                 float **d_rstd, float **d_inp, float **d_weight,
                 float **d_bias, bool pinned = false) {

  cudaCheck(hipFree(*d_out));
  cudaCheck(hipFree(*d_mean));
  cudaCheck(hipFree(*d_rstd));
  cudaCheck(hipFree(*d_inp));
  cudaCheck(hipFree(*d_weight));
  cudaCheck(hipFree(*d_bias));
  if (pinned) {
    cudaCheck(hipHostFree(*out));
    cudaCheck(hipHostFree(*mean));
    cudaCheck(hipHostFree(*rstd));
    cudaCheck(hipHostFree(*inp));
    cudaCheck(hipHostFree(*weight));
    cudaCheck(hipHostFree(*bias));
  } else {
    free(*out);
    free(*mean);
    free(*rstd);
    free(*inp);
    free(*weight);
    free(*bias);
  }
}

void resetCPUMemory(float **out, float **mean, float **rstd, float **inp,
                    float **weight, float **bias) {
  free(*out);
  free(*mean);
  free(*rstd);
  free(*inp);
  free(*weight);
  free(*bias);
}

bool isPinnedMemory(int pinnedMemoryKernels[], int kernelNum, size_t N) {
  for (int i = 0; i < N; i++) {
    if (kernelNum == pinnedMemoryKernels[i])
      return true;
  }
  return false;
}

// GPU kernels

// naive drag and drop implementation into kernel, parallelize over B,T, loop
// over C
__global__ void layernorm_forward_kernel1(float *out, float *mean, float *rstd,
                                          const float *inp, const float *weight,
                                          const float *bias, int N, int C) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float eps = 1e-5f;

  if (idx < N) {
    // seek to the input position inp[idx,:]
    const float *x = inp + idx * C;
    // calculate the mean
    float m = 0.0f;
    for (int i = 0; i < C; i++) {
      m += x[i];
    }
    m = m / C;
    // calculate the variance (without any bias correction)
    float v = 0.0f;
    for (int i = 0; i < C; i++) {
      float xshift = x[i] - m;
      v += xshift * xshift;
    }
    v = v / C;
    // calculate the rstd
    float s = 1.0f / sqrtf(v + eps);
    // seek to the output position in out[idx,:]
    float *out_idx = out + idx * C;
    for (int i = 0; i < C; i++) {
      float n = (s * (x[i] - m));        // normalized output
      float o = n * weight[i] + bias[i]; // scale and shift it
      out_idx[i] = o;                    // write
    }
    // cache the mean and rstd for the backward pass later
    mean[idx] = m;
    rstd[idx] = s;
  }
}

__global__ void mean_kernel(float *mean, const float *inp, int N, int C,
                            int block_size) {
  extern __shared__ float shared[];
  int idx = blockIdx.x;  // range [0, B*T)
  int tid = threadIdx.x; // range [0, block_size)
  const float *x = inp + idx * C;
  // thread coarsening
  float sum = 0.0f;
  for (int i = tid; i < C; i += block_size) {
    sum += x[i];
  }
  shared[tid] = sum;
  __syncthreads();
  // reductions
  for (int stride = block_size / 2; stride >= 1; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      shared[tid] += shared[tid + stride];
    }
  }
  // write the final result (at thread 0) to global memory
  if (tid == 0) {
    mean[idx] = shared[0] / C;
  }
}

__global__ void rstd_kernel(float *rstd, const float *inp, const float *mean,
                            int N, int C, int block_size) {
  extern __shared__ float shared[];
  int idx = blockIdx.x;  // range [0, B*T)
  int tid = threadIdx.x; // range [0, block_size)
  const float *x = inp + idx * C;
  float m = mean[idx];
  // thread coarsening
  float sum = 0.0f;
  for (int i = tid; i < C; i += block_size) {
    float diff = x[i] - m;
    sum += diff * diff;
  }
  shared[tid] = sum;
  __syncthreads();
  // reductions
  for (int stride = block_size / 2; stride >= 1; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      shared[tid] += shared[tid + stride];
    }
  }
  // write the final result (at thread 0) to global memory
  if (tid == 0) {
    rstd[idx] = 1.0f / sqrtf(shared[0] / C + 1e-5f);
  }
}

__global__ void normalization_kernel(float *out, const float *inp, float *mean,
                                     float *rstd, const float *weight,
                                     const float *bias, int B, int T, int C) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int bt = idx / C;
  int c = idx % C;

  float m = mean[bt];
  float s = rstd[bt];
  float xi = inp[idx];
  float n = s * (xi - m);
  float o = n * weight[c] + bias[c];

  out[idx] = o;
}

__global__ void layernorm_forward_kernel3(
    float *__restrict__ out, float *__restrict__ mean, float *__restrict__ rstd,
    const float *__restrict__ inp, const float *__restrict__ weight,
    const float *__restrict__ bias, int N, int C) {
  namespace cg = cooperative_groups;
  cg::thread_block block = cg::this_thread_block();
  cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
  // meta_group_size is the number of warps in a block, and meta_group_rank is
  // the warp index
  int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
  if (idx >= N) {
    return;
  }

  // the row of input that this group of threads is responsible for
  const float *x = inp + idx * C;

  // mean
  float sum = 0.0f;
  for (int i = warp.thread_rank(); i < C; i += warp.size()) {
    sum += x[i];
  }
  sum = cg::reduce(warp, sum, cg::plus<float>{});
  float m = sum / C;
  if (warp.thread_rank() == 0 && mean != nullptr) {
    __stcs(mean + idx, m);
  }

  // rstd
  sum = 0.0f;
  for (int i = warp.thread_rank(); i < C; i += warp.size()) {
    float diff = x[i] - m;
    sum += diff * diff;
  }
  sum = cg::reduce(warp, sum, cg::plus<float>{});
  float s = rsqrtf(sum / C + 1e-5f);
  if (warp.thread_rank() == 0 && rstd != nullptr) {
    __stcs(rstd + idx, s);
  }

  // final normalization and scaling by weight/bias
  float *o = out + idx * C;
  for (int c = warp.thread_rank(); c < C; c += warp.size()) {
    // load and store using the .cs "streaming" hint to the compiler,
    // indicating that this data will not be reused soon, and can be streamed
    // through the caches this allows the threads to get more cache-hits for the
    // (shared) weight and bias parameters
    float n = s * (__ldcs(x + c) - m);
    __stcs(o + c, n * weight[c] + bias[c]);
  }
}

// same as kernel 3 but uses var(x) == mean(x**2) - mean(x)**2
__global__ void layernorm_forward_kernel4(
    float *__restrict__ out, float *__restrict__ mean, float *__restrict__ rstd,
    const float *__restrict__ inp, const float *__restrict__ weight,
    const float *__restrict__ bias, int N, int C) {
  namespace cg = cooperative_groups;
  cg::thread_block block = cg::this_thread_block();
  cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
  int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
  if (idx >= N) {
    return;
  }

  // the row of input that this group of threads is responsible for
  const float *x = inp + idx * C;

  // thread coarsening through the row, reduce the sum in series
  float sum = 0.0;  // stores sum(x)
  float sum2 = 0.0; // stores sum(x**2)
  for (int i = warp.thread_rank(); i < C; i += warp.size()) {
    float xi = x[i];
    sum += xi;
    sum2 += xi * xi;
  }
  // warp-level reduction at the end
  sum = cg::reduce(warp, sum, cg::plus<float>{});   // sum(x)
  sum2 = cg::reduce(warp, sum2, cg::plus<float>{}); // sum(x**2)
  sum /= C;                                         // mean(x)
  sum2 /= C;                                        // mean(x**2)

  // mean, var, rstd
  float m = sum;
  float var = sum2 - sum * sum;
  float s = rsqrtf(var + 1e-5f);

  // store the mean, no need to cache it
  if (warp.thread_rank() == 0 && mean != nullptr) {
    __stcs(mean + idx, m);
  }
  // store the rstd, no need to cache it
  if (warp.thread_rank() == 0 && rstd != nullptr) {
    __stcs(rstd + idx, s);
  }
  // final normalization and scaling by weight/bias
  float *o = out + idx * C;
  for (int c = warp.thread_rank(); c < C; c += warp.size()) {
    float n = s * (__ldcs(x + c) - m);
    __stcs(o + c, n * weight[c] + bias[c]);
  }
}

// like 4, but in kernel 5 we have each block doing one row, not just a single
// warp
__global__ void layernorm_forward_kernel5(
    float *__restrict__ out, float *__restrict__ mean, float *__restrict__ rstd,
    const float *__restrict__ inp, const float *__restrict__ weight,
    const float *__restrict__ bias, int N, int C) {
  namespace cg = cooperative_groups;
  cg::thread_block block = cg::this_thread_block();
  cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
  __shared__ float shared_sum[32];  // block_size max is 1024 = 32 * 32 warps
  __shared__ float shared_sum2[32]; // warps will be writing into shared memeory
                                    // after warp-reduce
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int lane_id = threadIdx.x % 32;
  int idx = blockIdx.x; // simpoy one block per row
  // the row of input that this group of threads is responsible for
  const float *x = inp + idx * C;
  // thread coarsening through the row, reduce the sum in series
  float thread_sum = 0.0;  // stores sum(x)
  float thread_sum2 = 0.0; // stores sum(x**2)
  // for (int i = C + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
    float xi = x[i];
    thread_sum += xi;
    thread_sum2 += xi * xi;
  }
  // warp-level reduction
  float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{}); // sum(x)
  float warp_sum2 =
      cg::reduce(warp, thread_sum2, cg::plus<float>{}); // sum(x**2)
  // store the warp-level reduction in shared memory (we could have lane_id == 0
  // guard but not needed)
  shared_sum[warp_id] = warp_sum;
  shared_sum2[warp_id] = warp_sum2;
  __syncthreads();
  // load results from shared memory to threads, pad with zeros for threads that
  // are out of bounds
  warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
  warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
  // now reduce the warp-level reductions
  float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{}); // sum(x)
  float block_sum2 =
      cg::reduce(warp, warp_sum2, cg::plus<float>{}); // sum(x**2)
  // mean, var, rstd
  block_sum /= C;  // mean(x)
  block_sum2 /= C; // mean(x**2)
  float m = block_sum;
  float var = block_sum2 - m * m;
  float s = rsqrtf(var + 1e-5f);
  // store the mean, no need to cache it
  if (threadIdx.x == 0 && mean != nullptr) {
    __stcs(mean + idx, m);
  }
  // store the rstd, no need to cache it
  if (threadIdx.x == 0 && rstd != nullptr) {
    __stcs(rstd + idx, s);
  }
  // final normalization and scaling by weight/bias
  float *o = out + idx * C;
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
    float n = s * (__ldcs(x + i) - m);
    __stcs(o + i, n * weight[i] + bias[i]);
  }
}

// ----------------------------------------------------------------------------
// kernel launcher
void layernorm_forward1(float *out, float *mean, float *rstd, const float *inp,
                        const float *weight, const float *bias, int B, int T,
                        int C, const int block_size) {
  const int N = B * T;
  const int grid_size = ceil_div(N, block_size);
  layernorm_forward_kernel1<<<grid_size, block_size>>>(out, mean, rstd, inp,
                                                       weight, bias, N, C);
  cudaCheck(hipGetLastError());
}

void layernorm_forward2(float *out, float *mean, float *rstd, const float *inp,
                        const float *weight, const float *bias, int B, int T,
                        int C, const int block_size) {
  int N = B * T;
  // in mean and rstd, threads cooperate within blocks via reductions
  mean_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(mean, inp, N,
                                                                 C, block_size);
  cudaCheck(hipGetLastError());
  rstd_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(
      rstd, inp, mean, N, C, block_size);
  cudaCheck(hipGetLastError());
  // in the normalization, everything just gets flattened out
  const int block_size2 = 256;
  const int grid_size = ceil_div(B * T * C, block_size2);
  normalization_kernel<<<grid_size, block_size2>>>(out, inp, mean, rstd, weight,
                                                   bias, B, T, C);
  cudaCheck(hipGetLastError());
}

void layernorm_forward3(float *out, float *mean, float *rstd, const float *inp,
                        const float *weight, const float *bias, int B, int T,
                        int C, const int block_size) {
  assert(block_size % 32 == 0);
  const int N = B * T;
  const int grid_size = ceil_div(N * 32, block_size);
  layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp,
                                                       weight, bias, N, C);
  cudaCheck(hipGetLastError());
}

void layernorm_forward4(float *out, float *mean, float *rstd, const float *inp,
                        const float *weight, const float *bias, int B, int T,
                        int C, const int block_size) {
  assert(block_size % 32 == 0);
  const int N = B * T;
  const int grid_size = ceil_div(N * 32, block_size);
  layernorm_forward_kernel4<<<grid_size, block_size>>>(out, mean, rstd, inp,
                                                       weight, bias, N, C);
  cudaCheck(hipGetLastError());
}

void layernorm_forward5(float *out, float *mean, float *rstd, const float *inp,
                        const float *weight, const float *bias, int B, int T,
                        int C, const int block_size) {
  assert(block_size % 32 == 0);
  const int N = B * T;
  const int grid_size = N;
  layernorm_forward_kernel5<<<grid_size, block_size>>>(out, mean, rstd, inp,
                                                       weight, bias, N, C);
  cudaCheck(hipGetLastError());
}

void layernorm_forward6(float *d_out, float *d_mean, float *d_rstd,
                        float *d_inp, float *d_weight, float *d_bias,
                        float *inp, float *weight, float *bias, int B, int T,
                        int C, const int block_size, hipStream_t *streams,
                        int nStreams) {
  const int nChunk = 64;
  const int N = nChunk * T;
  size_t sToken = C * sizeof(float);
  const int grid_size = ceil_div(N, block_size);

  cudaCheck(hipGetLastError());

  cudaCheck(hipMemcpyAsync(d_weight, weight, sToken, hipMemcpyHostToDevice,
                            streams[nStreams - 1]));
  cudaCheck(hipMemcpyAsync(d_bias, bias, sToken, hipMemcpyHostToDevice,
                            streams[nStreams - 1]));

  for (int b = 0, sNum = 0; b < B; b += nChunk, sNum = (sNum + 1) % nStreams) {
    cudaCheck(hipMemcpyAsync(d_inp, inp, N * sToken, hipMemcpyHostToDevice,
                              streams[sNum]));
    layernorm_forward_kernel1<<<grid_size, block_size, 0, streams[sNum]>>>(
        d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, N, C);

    d_out = d_out + N * C;
    d_mean = d_mean + N;
    d_rstd = d_rstd + N;
    d_inp = d_inp + N * C;
    inp = inp + N * C;
  }

  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
}

// kernel version dispatch
void layernorm_forward(int kernel_num, float *d_out, float *d_mean,
                       float *d_rstd, float *d_inp, float *d_weight,
                       float *d_bias, float *out, float *mean, float *rstd,
                       float *inp, float *weight, float *bias, int B, int T,
                       int C, const int block_size, int nStreams,
                       hipStream_t *streams = nullptr) {

  switch (kernel_num) {
  case 1:
    layernorm_forward1(d_out, d_mean, d_rstd, d_inp, weight, bias, B, T, C,
                       block_size);
    break;
  case 2:
    layernorm_forward2(d_out, d_mean, d_rstd, d_inp, weight, bias, B, T, C,
                       block_size);
    break;
  case 3:
    layernorm_forward3(d_out, d_mean, d_rstd, d_inp, weight, bias, B, T, C,
                       block_size);
    break;
  case 4:
    layernorm_forward4(d_out, d_mean, d_rstd, d_inp, weight, bias, B, T, C,
                       block_size);
    break;
  case 5:
    layernorm_forward5(d_out, d_mean, d_rstd, d_inp, weight, bias, B, T, C,
                       block_size);
    break;
  case 6:
    layernorm_forward6(d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, inp,
                       weight, bias, B, T, C, block_size, streams, nStreams);
    break;
  default:
    printf("Invalid kernel number\n");
    exit(1);
  }
}

int main(int argc, char **argv) {
  srand(0);

  int B = 256;
  int T = 1024;
  int C = 768;

  int deviceIdx = 0;
  cudaCheck(hipSetDevice(deviceIdx));

  // host pointers
  float *out = nullptr;
  float *mean = nullptr;
  float *rstd = nullptr;
  float *inp = nullptr;
  float *weight = nullptr;
  float *bias = nullptr;

  // reference implementation pointers
  float *rOut = nullptr;
  float *rMean = nullptr;
  float *rRstd = nullptr;
  float *rInp = nullptr;
  float *rWeight = nullptr;
  float *rBias = nullptr;

  // device pointers
  float *d_out = nullptr;
  float *d_mean = nullptr;
  float *d_rstd = nullptr;
  float *d_inp = nullptr;
  float *d_weight = nullptr;
  float *d_bias = nullptr;

  // read kernel_num from command line
  int kernel_num = 2;
  if (argc > 1) {
    kernel_num = atoi(argv[1]);
  }
  printf("Using kernel %d\n", kernel_num);

  prepareCPUMemory(&rOut, &rMean, &rRstd, &rInp, &rWeight, &rBias, B, T, C);
  layernorm_forward_cpu(rOut, rMean, rRstd, rInp, rWeight, rBias, B, T, C);

  int block_sizes[] = {32, 64, 128, 256, 512, 1024};
  int pinned_memory_kernels[1] = {6};
  bool pinned = isPinnedMemory(pinned_memory_kernels, kernel_num,
                               sizeof(pinned_memory_kernels) / sizeof(int));

  const int nStreams = 8;
  hipStream_t streams[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipStreamCreate(&streams[i]);
  }

  // check the correctness of the kernel at all block sizes
  prepareMemory(&out, &mean, &rstd, &inp, &weight, &bias, &d_out, &d_mean,
                &d_rstd, &d_inp, &d_weight, &d_bias, B, T, C, pinned);
  for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
    int block_size = block_sizes[j];
    printf("Checking block size %d.\n", block_size);

    layernorm_forward(kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight,
                      d_bias, out, mean, rstd, inp, weight, bias, B, T, C,
                      block_size, nStreams, streams);

    validate_result(d_out, rOut, "out", B * T * C, 1e-5f);
    validate_result(d_mean, rMean, "mean", B * T, 1e-5f);
    validate_result(d_rstd, rRstd, "rstd", B * T, 1e-5f);
  }

  printf("All results match. Starting benchmarks.\n\n");

  // time the kernel at different block sizes
  for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
    int block_size = block_sizes[j];

    int repeat_times = 2;
    float elapsed_time = benchmark_kernel(
        repeat_times, layernorm_forward, kernel_num, d_out, d_mean, d_rstd,
        d_inp, d_weight, d_bias, out, mean, rstd, inp, weight, bias, B, T, C,
        block_size, nStreams, streams);

    // napkin math: estimate the memory bandwidth achieved
    // e.g. A100 40GB PCIe is advertised at 1,555GB/s
    long memory_ops = (2 * B * T * C) * 4; // *4 for float
    float memory_bandwidth = memory_ops / elapsed_time / 1e6;

    printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size,
           elapsed_time, memory_bandwidth);
  }

  for (int i = 0; i < nStreams; i++) {
    hipStreamDestroy(streams[i]);
  }
  resetMemory(&out, &mean, &rstd, &inp, &weight, &bias, &d_out, &d_mean,
              &d_rstd, &d_inp, &d_weight, &d_bias, pinned);
  resetCPUMemory(&rOut, &rMean, &rRstd, &rInp, &rWeight, &rBias);

  return 0;
}
