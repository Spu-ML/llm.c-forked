#include "hip/hip_runtime.h"
#define TESTING
#include <string.h>
#include "train_gpt2.cu"

// poor man's tensor checker
int check_tensor(float *a, float *b, int n, char* label) {
    int print_upto = 5;
    int ok = 1;
    printf("%s\n", label);
    for (int i = 0; i < n; i++) {
        if (fabsf(a[i] - b[i]) <= 1e-2) {
            if (i < print_upto) { printf("OK "); }
        } else {
            if (i < print_upto) { printf("NOT OK "); }
            ok = 0;
        }
        if (i < print_upto) { printf("%f %f\n", a[i], b[i]); }
    }
    // print the final result
    if (ok) {
        printf("TENSOR OK\n");
    } else {
        printf("TENSOR NOT OK\n");
    }
    return ok;
}

int check_decoder() {
    char decoder[GPT2_NUM_TOKENS][GPT2_MAX_TOKEN_LEN];
    gpt2_load_decoder("data/decode_gpt2.txt", decoder);

    static int tokens[] = {50256, 16773, 18162, 21986, 11, 198, 13681, 263, 23875, 198, 3152, 262, 11773, 2910, 198, 1169, 6002, 6386, 2583, 286, 262, 11858, 198, 20424, 428, 3135, 7596, 995, 3675, 13, 198, 40, 481, 407, 736, 17903, 11, 329, 703, 6029, 706, 4082, 198, 42826, 1028, 1128, 633, 263, 11, 198, 10594, 407, 198, 2704, 454, 680, 1028, 262, 1027, 28860, 286, 198, 3237, 323};
    static char* expected[] = {"<|endoftext|>", "Come", " Running", " Away", ",", "\n", "Great", "er", " conquer", "\n", "With", " the", " Imperial", " blood", "\n", "the", " heav", "iest", " host", " of", " the", " gods", "\n", "into", " this", " wond", "rous", " world", " beyond", ".", "\n", "I", " will", " not", " back", " thee", ",", " for", " how", " sweet", " after", " birth", "\n", "Netflix", " against", " rep", "ound", "er", ",", "\n", "will", " not", "\n", "fl", "our", "ish", " against", " the", " ear", "locks", " of", "\n", "All", "ay"};
    int num = sizeof(tokens) / sizeof(tokens[0]);

    int ok = 1;
    for (int i = 0; i < num; ++i) {
        if (strcmp(decoder[tokens[i]], expected[i]) != 0) {
            printf("MISMATCH AT INDEX %d: %s %s\n", i, decoder[tokens[i]], expected[i]);
            ok = 0;
        }
    }
    if (ok) {
        printf("Decoder OK\n");
    } else {
        printf("Decoder NOT OK\n");
    }
    return ok;
}

int main(int argc, char *argv[]) {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    enable_tf32 = 0; // NOTE: disable TF32 for testing!!!
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    int decoder_ok = check_decoder();
    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");

    int C = model.config.channels;
    int V = model.config.vocab_size;
    int maxT = model.config.max_seq_len;
    int L = model.config.num_layers;

    // load additional information that we will use for debugging and error checking
    FILE *state_file = fopen("gpt2_124M_debug_state.bin", "rb");
    if (state_file == NULL) { printf("Error opening state file\n"); exit(1); }
    int state_header[256];
    fread(state_header, sizeof(int), 256, state_file);
    if (state_header[0] != 20240327) { printf("Bad magic state file"); exit(1); }
    if (state_header[1] != 1) { printf("Bad version in state file"); exit(1); }
    int B = state_header[2]; // batch size, e.g. 4
    int T = state_header[3]; // time / sequence length (e.g. 64, up to maxT)
    printf("[State]\n");
    printf("batch_size: %d\n", B);
    printf("seq_len: %d\n", T);

    ParameterTensors expected_grads;
    float* expected_grads_memory = malloc_and_point_parameters(&expected_grads, model.param_sizes, 0);

    // inputs and expected outputs, only used for error checking
    int* x = (int*) malloc(B * T * sizeof(int));
    int* y = (int*) malloc(B * T * sizeof(int));
    float* expected_logits = (float*) malloc(B * T * V * sizeof(float));
    float* expected_loss = (float*) malloc(1 * sizeof(float));

    // read reference information from Python
    fread(x, sizeof(int), B*T, state_file);
    fread(y, sizeof(int), B*T, state_file);
    fread(expected_logits, sizeof(float), B*T*V, state_file);
    fread(expected_loss, sizeof(float), 1, state_file);
    fread(expected_grads_memory, sizeof(float), model.num_parameters, state_file);
    fclose(state_file);

    // overall OK signal for the test
    int allok = decoder_ok;

    // let's do 10 training iterations, following the pytorch code
    float losses[10];
    for (int step = 0; step < 10; step++) {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);
        gpt2_forward(&model, x, y, B, T);
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

        if (step == 0) {
            // error checking at step 0 for reference activations

            // at this point, target should be equal to expected_logits, let's compare
            // copy logits to CPU so we can compare them
            float* logits_cpu = (float*) malloc(B * T * V * sizeof(float));
            hipMemcpy(logits_cpu, model.acts.logits, B * T * V * sizeof(float), hipMemcpyDeviceToHost);
            int logits_ok = 1;
            for (int i=0; i<B*T*V; i++) {
                if(i < 3) {
                    printf("%f %f\n", expected_logits[i], logits_cpu[i]);
                }
                if (fabsf(expected_logits[i] - logits_cpu[i]) >= 1e-2) {
                    printf("MISMATCH AT INDEX %d: ", i);
                    printf("%f %f\n", expected_logits[i],logits_cpu[i]);
                    logits_ok = 0;
                    break;
                }
            }
            if(!logits_ok) { printf("NOT "); }
            printf("OK (LOGITS)\n");
            allok = allok && logits_ok;
            free(logits_cpu);

            // compare the achieved loss
            if (fabsf(model.mean_loss - *expected_loss) >= 1e-2) {
                printf("LOSS MISMATCH: %f %f\n", model.mean_loss, *expected_loss);
                allok = 0;
            } else {
                printf("LOSS OK: %f %f\n", model.mean_loss, *expected_loss);
            }
        }
    }

    printf("overall okay: %d\n", allok);

    // free everything
    free(x);
    free(y);
    free(expected_logits);
    free(expected_loss);
    free(expected_grads_memory);
    gpt2_free(&model);
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));

    return 0;
}
